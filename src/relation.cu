#include "hip/hip_runtime.h"
#include "relation.h"
#include <cstdio>
#include "util.h"

__global__ void print_gpu_kernel(Tuple* data, int count) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < count) {
        Tuple tuple = data[idx];
        printf("(%d, %d)\n", tuple.x, tuple.y);
    }
}

Relation::Relation(Tuple* tuples, int numberTuples) {
    data = tuples;
    count = numberTuples;
}

Relation::Relation() { }

void Relation::free() {
    CUDA_CHECK(hipFree(data));
}

void Relation::print_gpu() {
    printf("------------ (Relation)\n");
    int blockSize = 256;
    int blocks = (count + blockSize - 1) / blockSize;
    print_gpu_kernel<<<blocks, blockSize>>>(data, count);
    hipDeviceSynchronize();
    printf("------------ (%d tuples)\n", count);
}

void Relation::print_stats() {
    printf("%d tuples\n", count);
}

Relation Relation::transferToDevice() {
    Relation deviceRelation;
    CUDA_CHECK(hipMalloc(&deviceRelation.data, count * sizeof(Tuple)));
    CUDA_CHECK(hipMemcpy(deviceRelation.data, data, count * sizeof(Tuple), hipMemcpyHostToDevice));
    deviceRelation.count = count;
    return deviceRelation;
}