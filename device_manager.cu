#include "hip/hip_runtime.h"
#include "device_manager.h"
#include <stdio.h>

__global__ void cuda_echo() {
    printf("GPU Test.\n");
}

__global__ void cuda_print(Relation relation, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto [x, y] = relation[idx];
    if (idx < n) {
        printf("%d: (%d, %d)\n", idx, x, y);
    }
}

Relation DeviceManager::TransferDataToDevice(Dataset* ds) {
    Relation deviceArray;
    hipMalloc(&deviceArray, ds->size_bytes());
    hipMemcpy(deviceArray, ds->relation(), ds->size_bytes(), hipMemcpyHostToDevice);
    relations.push_back(deviceArray);
    return deviceArray;
}

void DeviceManager::Echo() {
    cuda_echo<<<1,1>>>();
    hipDeviceSynchronize(); 
}

void DeviceManager::PrintRelation(Tuple* relation, int maxCount) {
    cuda_print<<<1,32>>>(relation, maxCount);
    hipDeviceSynchronize(); 
}

DeviceManager::~DeviceManager() {
    for (auto& relation: relations) {
        hipFree(relation);
    }
}
